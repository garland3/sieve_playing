#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

__global__ void sieve_kernel(bool* is_prime, int start, int end) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = 2 + tid; i * i <= end; i += step) {
        for (int j = max(2, (start + i - 1) / i) * i; j <= end; j += i) {
            is_prime[j - start] = false;
        }
    }
}

int* sieve(int n, int& num_primes) {
    int chunk_size = 1000000;
    int num_chunks = (n + chunk_size - 1) / chunk_size;
    int* primes = new int[n];
    num_primes = 0;
    for (int i = 0; i < num_chunks; i++) {
        int start = i * chunk_size + 2;
        int end = min(start + chunk_size - 1, n);
        int size = end - start + 1;
        bool* is_prime = new bool[size];
        for (int j = 0; j < size; j++) {
            is_prime[j] = true;
        }

        bool* d_is_prime;
        hipMalloc(&d_is_prime, size * sizeof(bool));
        hipMemcpy(d_is_prime, is_prime, size * sizeof(bool), hipMemcpyHostToDevice);

        int threads_per_block = 256;
        int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
        sieve_kernel<<<blocks_per_grid, threads_per_block>>>(d_is_prime, start, end);

        hipMemcpy(is_prime, d_is_prime, size * sizeof(bool), hipMemcpyDeviceToHost);
        hipFree(d_is_prime);

        for (int j = 0; j < size; j++) {
            if (is_prime[j]) {
                primes[num_primes] = start + j;
                num_primes++;
            }
        }
        delete[] is_prime;
    }
    return primes;
}

int main() {
    int n = 100000000;

    auto start_time = high_resolution_clock::now();
    int num_primes;
    int* primes = sieve(n, num_primes);
    auto end_time = high_resolution_clock::now();

    auto duration = duration_cast<milliseconds>(end_time - start_time);
    cout << "Execution time: " << duration.count() << " milliseconds" << endl;
    cout << "Number of primes found: " << num_primes << endl;

    delete[] primes;

    return 0;
}
